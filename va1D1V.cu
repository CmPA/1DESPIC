#include "hip/hip_runtime.h"
/** @file va1D1V.cpp
 *  @brief Vlasov-Ampere 1D electrostatic particle-in-cell code for GPU
 *  @author Diego GONZALEZ-HERRERO <diego.gonzalez@kuleuven.be>
 *  @date June, 2016
 *
 *   Copyright (c) 2016 KU Leuven University
 *   Some rights reserved. See COPYING, AUTHORS.
 *
 *  @license GPL-3.0 <https://opensource.org/licenses/GPL-3.0>
 */

////////////////////////////////////////////////////////////////////////////////
////                                                                        ////
////  compile with CUDA3.2: nvcc -arch=sm_21 main.cu                        ////
////                                                                        ////
////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <ctime>
#include "histogram.h"

#define NUMBER float

#define WITH_PART_IO

typedef struct {
  char   name[256];
  int    nc;
  int    nppc;
  int    cycles;
  NUMBER dt;
  NUMBER dx;
  NUMBER rho;
  NUMBER qom;
  NUMBER vth;
  NUMBER v0;
  int    ioc;
  } InputData;


 __device__ inline void atomicAdd (double *address, double value)
{
  unsigned long long oldval, newval, readback;

  oldval = __double_as_longlong(*address);
  newval = __double_as_longlong(__longlong_as_double(oldval) + value);
  while ((readback=atomicCAS((unsigned long long *)address, oldval, newval)) != oldval)
  {
    oldval = readback;
    newval = __double_as_longlong(__longlong_as_double(oldval) + value);
  }
}

// <<1, NumNodes>>
__global__ void set_current( NUMBER *jx, int nodThre)
{
//if ((threadIdx.x == 0) && (blockIdx.x == 0)) printf("set current\n");
  int tid = (threadIdx.x + blockIdx.x * blockDim.x)*nodThre;
  for (int i=0; i<nodThre; i++, tid++) jx[tid] = 0;
}

// <<NumCell, threads>>
__global__ void add_current(NUMBER *Jx, NUMBER *rx, NUMBER *vx, NUMBER dx, NUMBER q, int parThre)
{
//if ((threadIdx.x == 0) && (blockIdx.x == 0)) printf("Add current\n");
  // Particle index
  int tid = (threadIdx.x + blockIdx.x * blockDim.x)*parThre;
  // Indes of last node:
  int lastN = gridDim.x;
  for (int i=0; i<parThre; i++, tid++) {
    NUMBER x = rx[tid];
    NUMBER v = vx[tid];

    // The particle has to be allways between the first and the last node
    int idc1 = int(x/dx);  // Cell on the left of the particle
    int idc2 = idc1+1;           // Cell of the right of the particle
    NUMBER x_dx= x/dx;
    NUMBER w1 = x_dx - idc1;
    NUMBER w2 = idc2 - x_dx;
  
    NUMBER qv = q*v/2.0;
    NUMBER val1 = qv*w1;
    NUMBER val2 = qv*w2;
  
    atomicAdd(&Jx[idc1], val1);    // sm_20 or more
    atomicAdd(&Jx[idc2], val2);    // sm_20 or more
  
    if (idc1 == 0) atomicAdd(&Jx[lastN], val1);
    if (idc2 == lastN) atomicAdd(&Jx[0], val2);
  }	
}
// <<1, NumNodes>>
__global__ void calc_field( NUMBER *Jx, NUMBER *Ex, NUMBER dt, int nodThre)
{
//if ((threadIdx.x == 0) && (blockIdx.x == 0)) printf("Calc field\n");
  int tid = (threadIdx.x + blockIdx.x * blockDim.x)*nodThre;
  for (int i=0; i<nodThre; i++, tid++) Ex[tid] -= Jx[tid]*dt;
}

// <<NumCell, threads>>
__global__ void update_part(NUMBER *rx, NUMBER *vx, NUMBER *Ex, NUMBER lbox, NUMBER qm, NUMBER dt, 
                           NUMBER dx, int parThre)
{
//if ((threadIdx.x == 0) && (blockIdx.x == 0)) printf("Update part\n");
  int tid = (threadIdx.x + blockIdx.x * blockDim.x)*parThre;

  for (int i=0; i<parThre; i++, tid++) {
    NUMBER x = rx[tid];
    NUMBER v = vx[tid];

    // Interpolate the electric field
    int idc1 = int(x/dx);  // Cell on the left of the particle
    int idc2 = idc1+1;           // Cell of the right of the particle
    NUMBER x_dx= x/dx;
    NUMBER w1 = x_dx - idc1;
    NUMBER w2 = idc2 - x_dx;
    NUMBER E = Ex[idc1]*w1 + Ex[idc2]*w2;
   
    // Update the velocity
    v += qm*E*dt;

    // Update the position
    x = x + v*dt;
    while (x < 0.0) x = x + lbox;
    while (x >= lbox) x = x - lbox;

    rx[tid] = x;
    vx[tid] = v;
  }
}


NUMBER Eenergy(NUMBER *Ex, int n, NUMBER dx)
{
  NUMBER en = 0;
  NUMBER fac = dx/(8*M_PI);
  for (int i=0; i<n; i++) {
    en += fac*Ex[i]*Ex[i];
  }
  return en;
}

NUMBER Kenergy(NUMBER *vx, int n)
{
  NUMBER en = 0;
  for (int i=0; i<n; i++) {
    en += 0.5*vx[i]*vx[i];
  }
  return en;
}


void recordHis(Histogram H, NUMBER* data, int n, const char* name)
{
  for (int i=0; i<n; i++) 
    H.count(data[i]);

  FILE* fd = fopen(name, "w");

  fprintf(fd,"# nd = %d \n", H.nd);
  fprintf(fd,"#H0: %d \n", H.H0);
  for (int i=0;i<H.nc;i++) {
    fprintf(fd, "%10.3e %13.6e \n",H.x0 + H.dx*(i+0.5),((NUMBER)H.H[i]/(H.nd*H.dx)));
  }
  fprintf(fd,"#H1: %d \n", H.H1);

  fclose(fd);
}


InputData readInput(const char* file) 
{
  FILE* fd = fopen(file,"r");
  if (NULL == fd) {
    fprintf(stderr,"ERROR: file %s cannot be opened for reading\n\n",file);
    exit(-1);
  }
  printf("Reading %s \n",file);

  InputData data;

  char line[256];
  char *txt;
  
  strncpy(line, file, sizeof(line));
  txt = strtok(line, "."); 
  strncpy(data.name, txt, sizeof(data.name));
  
  // First line not used
  fgets(line, sizeof(line), fd);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.nc = atoi(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.nppc = atoi(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.cycles = atoi(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.dt = atof(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.dx = atof(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.rho = atof(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.qom = atof(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.vth = atof(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.v0 = atof(txt);

  fgets(line, sizeof(line), fd);
  txt = strtok(line, ":"); txt = strtok(NULL, ":");
  data.ioc = atoi(txt);

  fclose(fd);

  return data;
}

////////////////////////////////////////////////////////////////////////////////
////
////    MAIN 
////
////////////////////////////////////////////////////////////////////////////////
////
////  Input file format: 
////           # First line as a comment
////           Number of cells              : 256
////           Number of particles per cell : 1000
////           Number of cycles             : 8000
////           Time step (dt)               : 0.1
////           Cell size (dx)               : 0.04
////           Charge density               : 1.0
////           Charge mass ratio (qom)      : -1.0
////           Thermal velocity             : 0.02
////           Drift velocity               : 0.1
////           IO cycles                    : 1000
////
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char* argv[]) 
{

  if (argc < 3) {
    fprintf(stderr,"\tUse %s <input file> <Threads per block>\n\n",argv[0]);
    exit(-1);
  }

  // Setup timer
  hipEvent_t start, stop;
  float cutime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  InputData data = readInput(argv[1]);
  int threads = atoi(argv[2]);

  int NumCell  = data.nc;
  int NumPart  = data.nppc;
  int Cycles   = data.cycles;
  int NumNodes = NumCell+1;

  if (0 != NumPart % threads) {
    fprintf(stderr,"ERROR: Number of particles per cell is not a multiple of the number of threads\n");
    exit(-1);
  }
  if (0 != NumNodes % threads) {
    fprintf(stderr,"ERROR: Number of nodes is not a multiple of the number of threads\n");
    exit(-1);
  }
 
  int ParThread = NumPart/threads;
  int NodThread = NumNodes/threads;

  // Histograms
  Histogram HisE;
  Histogram HisV;
  Histogram HisX;


  int TotPart = NumPart*NumCell;
  const NUMBER dt  = data.dt;
  const NUMBER dx  = data.dx;
  const NUMBER L   = NumCell*dx;
  const NUMBER rho = data.rho;

  const NUMBER qom = data.qom;
  const NUMBER v0  = data.v0;
  const NUMBER vth = data.vth;
  const NUMBER q   = (qom/fabs(qom))*(rho/NumPart)*dx;

  HisE.init(-1,1);
  HisV.init(-2*v0,2*v0);
  HisX.init(0,L);
  
  long long seed = 12313132;
  srand48(seed); 

  fprintf(stdout,"#######################################\n"
                 "#  GPU:  %s                            \n"
                 "#######################################\n"
                 " sizeof(NUMBER) = %d \n"
                 " Random numbers seed: %ld \n"
                 " Number of threads per block: %d\n"
                 "#######################################\n"
                 " Number of cells:           %d \n"
                 " Number of nodes:           %d \n"
                 " Particles per cell:        %d \n"
                 " Total number of particles: %d \n"
                 " Cycles:                    %d \n"
                 "#######################################\n"
                 " Time step dt = %g \n"
                 " Box size  L = %g \n"
                 " Delta X dx=%g\n"
                 " Charge/mass qom=%g\n"
                 " Charge q=%g\n"
                 " Initial velocity v0  = %g\n"
                 " Thermal velocity vth = %g\n"
                 "#######################################\n"
                 ,data.name
                 ,((int)sizeof(NUMBER))
                 ,seed
                 ,threads
                 ,NumCell
                 ,NumNodes
                 ,NumPart
                 ,TotPart
                 ,Cycles
                 ,dt
                 ,L
                 ,dx
                 ,qom
                 ,q
                 ,v0
                 ,vth
                 );
                  
  fprintf(stdout,"*** Memory allocation ***\n");

  size_t mem_tot  = 0;
  size_t mem_free = 0;
  hipMemGetInfo  (&mem_free, & mem_tot);
  fprintf(stdout,"Total memory: %.3f Gb    Free memory %.3f Gb\n", ((double)mem_tot/(1024*1024*1024)) , ((double)mem_free/(1024*1024*1024)));
  size_t memreq = 2*NumNodes*sizeof(NUMBER) + 2*TotPart*sizeof(NUMBER);
  fprintf(stdout,"Memory used: %.3f Gb \n", ((double)memreq)/(1024*1024*1024));

  /***********************************************/
  /*           Memory allocation                 */
  /***********************************************/
  // HOST:
  NUMBER h_Jx[NumNodes];
  NUMBER h_Ex[NumNodes];
  NUMBER h_vx[TotPart];
  NUMBER h_rx[TotPart];
  // DEVICE:
  NUMBER *d_Jx, *d_Ex, *d_rx, *d_vx;
  hipMalloc( (void**)&d_Jx, NumNodes*sizeof(NUMBER));
  hipMalloc( (void**)&d_Ex, NumNodes*sizeof(NUMBER));
  hipMalloc( (void**)&d_rx, TotPart*sizeof(NUMBER));
  hipMalloc( (void**)&d_vx, TotPart*sizeof(NUMBER));
	
  /***********************************************/
  /*     Initialization and copy to GPU          */
  /***********************************************/
  // Current and fields
  for (int i=0;i<NumNodes;i++) {
    h_Jx[i] = 0.0;
    h_Ex[i] = 0.0;
  }


  NUMBER sbp = L/(TotPart+1);

  // Position and velocity
  for (int i=0; i<TotPart;i++) {
#define RANDOM_SORT
#ifdef RANDOM_SORT
    h_rx[i] = drand48()*L;   // Random in all box
#else
    h_rx[i] = i*sbp;
#endif
    if (i % 2 == 0) h_vx[i] =   v0 + drand48()*vth;
    else            h_vx[i] = -(v0 + drand48()*vth);

//pow(-1.0,i)*(v0 + vt * (rand() % 1)); // base velocity + noise from thermal velocity
  }
 
  // Copy to the GPU
  hipMemcpy( d_Jx, h_Jx, NumNodes*sizeof(NUMBER), hipMemcpyHostToDevice);
  hipMemcpy( d_Ex, h_Ex, NumNodes*sizeof(NUMBER), hipMemcpyHostToDevice);
  hipMemcpy( d_rx, h_rx, TotPart*sizeof(NUMBER), hipMemcpyHostToDevice);
  hipMemcpy( d_vx, h_vx, TotPart*sizeof(NUMBER), hipMemcpyHostToDevice);

  /***********************************************/
  /*                 MAIN LOOP                   */
  /***********************************************/
  //Start kernel timer
  hipEventRecord(start, 0);

  char name[128];
  for (int c=0; c<Cycles; c++) {
	
    update_part<<<NumCell,threads>>>(d_rx, d_vx, d_Ex, L, qom, dt, dx, ParThread);	
    set_current<<<1, threads>>>(d_Jx, NodThread);
    add_current<<<NumCell,threads>>>(d_Jx, d_rx, d_vx, dx, q, ParThread);
    calc_field<<<1, threads>>>(d_Jx, d_Ex, dt, NodThread);

    if ((c) % data.ioc == 0) {
      printf("Cycle %d \n",c);
#if defined(WITH_HISTOGRAMS) || defined(WITH_PART_IO)
      hipMemcpy(h_rx, d_rx, TotPart*sizeof(NUMBER), hipMemcpyDeviceToHost);
      hipMemcpy(h_vx, d_vx, TotPart*sizeof(NUMBER), hipMemcpyDeviceToHost);
#endif

#ifdef WITH_HISTOGRAMS
      sprintf(name,"%s_rx_%d.txt",data.name, c);
      HisX.clean();
      recordHis(HisX, h_rx, TotPart, name);
      HisV.clean();
      sprintf(name,"%s_vx_%d.txt",data.name, c);
      recordHis(HisV, h_vx, TotPart, name);
#endif

#ifdef WITH_PART_IO
      sprintf(name,"%s_part_%d.txt",data.name, c);
      FILE* fd = fopen(name,"w");
      for (int i=0; i<TotPart; i++)
        fprintf(fd, "%13.6e %13.6e \n"
                ,h_rx[i], h_vx[i]);
      fclose(fd);
#endif
    }
  }

  /***********************************************/
  /*  Copy the results from GPU to the host      */
  /***********************************************/

  // copy the results back to the host memory
  hipMemcpy(h_Jx, d_Jx, NumNodes*sizeof(NUMBER), hipMemcpyDeviceToHost);
  hipMemcpy(h_Ex, d_Ex, NumNodes*sizeof(NUMBER), hipMemcpyDeviceToHost);
  hipMemcpy(h_rx, d_rx, TotPart*sizeof(NUMBER), hipMemcpyDeviceToHost);
  hipMemcpy(h_vx, d_vx, TotPart*sizeof(NUMBER), hipMemcpyDeviceToHost);
	
  // Stop kernel timer
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&cutime, start, stop);
  printf("Kernel execution time: %.3f s\n", cutime/1000);

#ifdef WITH_HISTOGRAMS
  sprintf(name,"%s_rx_%d.txt",data.name, Cycles);
  HisX.clean();
  recordHis(HisX, h_rx, TotPart, name);
  HisV.clean();
  sprintf(name,"%s_vx_%d.txt",data.name, Cycles);
  recordHis(HisV, h_vx, TotPart, name);
#endif

#ifdef WITH_PART_IO
  sprintf(name,"%s_part_%d.txt",data.name, Cycles);
  FILE* fd = fopen(name,"w");
  for (int i=0; i<TotPart; i++)
    fprintf(fd, "%13.6e %13.6e \n"
            ,h_rx[i], h_vx[i]);
  fclose(fd);
#endif

  // free the device memory
  hipFree(d_Jx);
  hipFree(d_Ex);
  hipFree(d_rx);
  hipFree(d_vx);


  // write output
  sprintf(name,"%s_Jx_GPU.txt",data.name);
  FILE* fd_Jx=fopen(name, "w");
  sprintf(name,"%s_Ex_GPU.txt",data.name);
  FILE* fd_Ex=fopen(name, "w");
	
  // write files
  for (int i=0;i<NumNodes;i++) {
    fprintf(fd_Jx, "%13.6e \n",h_Jx[i]);
    fprintf(fd_Ex, "%13.6e \n",h_Ex[i]);
  }
  // close files
  fclose(fd_Ex);
  fclose(fd_Jx);

  return(0);
} 
